#include "hip/hip_runtime.h"
#include <starpu.h>

static __global__ void cuda_dev_const(float *px, float k)
{
        int tid = threadIdx.x + blockIdx.x * blockDim.x;
        px[tid] = k;
}

static __global__ void cuda_dev_iota(float *px)
{
        int tid = threadIdx.x + blockIdx.x * blockDim.x;
        px[tid] = tid;
}

extern "C" void dev_const(void *descr[], void *_args)
{
        float *filt_data = (float *)STARPU_MATRIX_GET_PTR(descr[0]);

        int filt_size = STARPU_MATRIX_GET_NX(descr[0]);
        int filt_nb = STARPU_MATRIX_GET_NY(descr[0]);

        cuda_dev_const<<<filt_size, filt_nb, 0, starpu_cuda_get_local_stream()>>>(filt_data, 1.f);

        hipError_t status = hipGetLastError();
        if (status != hipSuccess) STARPU_CUDA_REPORT_ERROR(status);
        hipStreamSynchronize(starpu_cuda_get_local_stream());
}

extern "C" void dev_iota(void *descr[], void *_args)
{
        float *in_data = (float *)STARPU_MATRIX_GET_PTR(descr[0]);
        int in_size = STARPU_MATRIX_GET_NX(descr[0]);
        int in_nb = STARPU_MATRIX_GET_NY(descr[0]);
 
        cuda_dev_iota<<<in_size, in_nb, 0, starpu_cuda_get_local_stream()>>>(in_data);

        hipError_t status = hipGetLastError();
        if (status != hipSuccess) STARPU_CUDA_REPORT_ERROR(status);
        hipStreamSynchronize(starpu_cuda_get_local_stream());
}