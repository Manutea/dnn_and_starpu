#include <starpu.h>
#include "hipDNN.h"

struct cudnn_convolution_params
{
  // input
  int in_n, in_c, in_h, in_w;
  // filter
  int filt_k, filt_c, filt_h, filt_w;
  // convolution
  int pad_h, pad_w, str_h, str_w, dil_h, dil_w;
  // out
  int out_n, out_c, out_h, out_w;
  // workspace size
  size_t ws_size;
};

extern "C" void init_conv_cudnn_func(void *buffers[], void *_args)
{
  //Tensor In, Filter, Convolution params
  cudnn_convolution_params *prms = (cudnn_convolution_params *)_args;

  float *in_data    = (float *)STARPU_VECTOR_GET_PTR(buffers[0]);
  float *filt_data  = (float *)STARPU_VECTOR_GET_PTR(buffers[1]);
  float *out_data   = (float *)STARPU_VECTOR_GET_PTR(buffers[2]);

  hipdnnHandle_t cudnn;
  hipdnnCreate(&cudnn);

  //Tensor in
  hipdnnTensorDescriptor_t in_desc;
  hipdnnCreateTensorDescriptor(&in_desc);
  hipdnnSetTensor4dDescriptor(in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, prms->in_n, prms->in_c, prms->in_h, prms->in_w);

  //Filter
  hipdnnFilterDescriptor_t filt_desc;
  hipdnnCreateFilterDescriptor(&filt_desc);
  hipdnnSetFilter4dDescriptor(filt_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, prms->filt_k, prms->filt_c, prms->filt_h, prms->filt_w);

  //Convoluion
  hipdnnConvolutionDescriptor_t conv_desc;
  hipdnnCreateConvolutionDescriptor(&conv_desc);
  hipdnnSetConvolution2dDescriptor(conv_desc, prms->pad_h, prms->pad_w, prms->str_h, prms->str_w,
  prms->dil_h, prms->dil_w, HIPDNN_CONVOLUTION, HIPDNN_DATA_FLOAT);

  //Setup the output tensor and allocate the proper amount of memory prior to launch the actual convolution
  int out_n, out_c, out_h, out_w;
  hipdnnGetConvolution2dForwardOutputDim(conv_desc, in_desc, filt_desc, &prms->out_n, &prms->out_c, &prms->out_h, &prms->out_w);

  //Tensor out
  hipdnnTensorDescriptor_t out_desc;
  hipdnnCreateTensorDescriptor(&out_desc);
  hipdnnSetTensor4dDescriptor(out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, prms->out_n, prms->out_c, prms->out_h, prms->out_w);
  
  //This function attempts all algorithms available for hipdnnConvolutionForward().
  int n_returnedAlgo;  
  const int n_requestedAlgo = 10;
  hipdnnConvolutionFwdAlgoPerf_t fwd_algo_perf[n_requestedAlgo];
  hipdnnFindConvolutionForwardAlgorithm(cudnn, in_desc, filt_desc, conv_desc, out_desc, n_requestedAlgo, &n_returnedAlgo, fwd_algo_perf);

  //This function returns the amount of GPU memory workspace the user needs to allocate to be able to call hipdnnConvolutionForward() with the specified algorithm.
  hipdnnConvolutionFwdAlgo_t fwd_algo = fwd_algo_perf[0].algo;
  hipdnnGetConvolutionForwardWorkspaceSize(cudnn, in_desc, filt_desc, conv_desc, out_desc, fwd_algo, &prms->ws_size);

  //float *ws_data;
  //hipMalloc(&ws_data, prms_in->ws_size);

  // perform
  float alpha = 1.f;
  float beta  = 0.f;

  hipdnnConvolutionForward(cudnn, &alpha, in_desc, in_data, filt_desc, filt_data, conv_desc, fwd_algo, NULL/*ws_data*/, prms->ws_size, &beta, out_desc, out_data);

  // results  
  //std::cout << "in_data:" << std::endl;
  //print(in_data, in_n, in_c, in_h, in_w);
  //
  //std::cout << "filt_data:" << std::endl;
  //print(filt_data, filt_k, filt_c, filt_h, filt_w);
  //
  //std::cout << "out_data:" << std::endl;
  //print(out_data, out_n, out_c, out_h, out_w);

  // finalizing
  //hipdnnDestroyTensorDescriptor(out_desc);
  //hipdnnDestroyConvolutionDescriptor(conv_desc);
  //hipdnnDestroyFilterDescriptor(filt_desc);
  //hipdnnDestroyTensorDescriptor(in_desc);
  //hipdnnDestroy(cudnn);
}